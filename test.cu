#include "hip/hip_runtime.h"
#include <iostream>

#if defined XXCUDA
    #include "util_cuda.h"
#elif defined XXHIP
    #include "util_cuda.h"
#else
    #error "one of XXCUDA or XXHIP must #defined"
#endif


// read command line arguments
int read_arg(int argc, char** argv, int index, int default_value) {
    if(argc>index) {
        try {
            auto n = std::stoi(argv[index]);
            if(n<0) {
                return default_value;
            }
            return n;
        }
        catch (std::exception e) {
            std::cout << "error : invalid argument \'" << argv[index]
                      << "\', expected a positive integer." << std::endl;
            exit(1);
        }
    }

    return default_value;
}

namespace kernels {

__global__
void empty(unsigned n) {}

__global__
void axpy(double *y, const double* x, double alpha, unsigned n) {
    auto i = threadIdx.x + blockIdx.x*blockDim.x;

    if (i<n) {
        y[i] += alpha*x[i];
    }
}

__device__
double f(double x) {
    return exp(cos(x))-2;
};

__device__
double fp(double x) {
    return -sin(x) * exp(cos(x));
};

__global__
void newton(double *x, unsigned n) {
    auto i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i<n) {
        auto x0 = x[i];
        for(int iter=0; iter<5; ++iter) {
            x0 -= f(x0)/fp(x0);
        }
        x[i] = x0;
    }
}

}

int main(int argc, char** argv) {
    const unsigned pow = read_arg(argc, argv, 1, 20);
    const unsigned n = 2 << pow;
    const unsigned block_dim = read_arg(argc, argv, 2, 128);
    const unsigned grid_dim = (n-1)/block_dim + 1;

    std::cout << "n " << n << " blockdim " << block_dim << " griddim " << grid_dim << "\n";

    // Run the newton kernel a bunch of times on a larger array to "warm up"
    {
        unsigned ni = 2<<24;
        double* xhi = malloc_host<double>(ni);
        double* xi = malloc_device<double>(ni);
        std::fill(xhi, xhi+ni, 2.3);
        copy_to_device<double>(xhi, xi, ni);
        for (auto i=0; i<1000; ++i) {
            kernels::newton<<<grid_dim, block_dim>>>(xi, ni);
        }
        std::free(xhi);
        free_device(xi);
    }


    double* xh = malloc_host<double>(n);
    double* yh = malloc_host<double>(n);
    std::fill(xh, xh+n, 2.0);
    std::fill(yh, yh+n, 1.0);

    double* x = malloc_device<double>(n);
    double* y = malloc_device<double>(n);
    copy_to_device<double>(xh, x, n);
    copy_to_device<double>(yh, y, n);

    device_synch();
    start_gpu_prof();

    int nruns = 10;

    for (auto i=0; i<nruns; ++i) {
        kernels::newton<<<grid_dim, block_dim>>>(x, n);
    }

    for (auto i=0; i<nruns; ++i) {
        kernels::axpy<<<grid_dim, block_dim>>>(y, x, 2.0, n);
    }

    for (auto i=0; i<nruns; ++i) {
        kernels::empty<<<grid_dim, block_dim>>>(n);
    }

    stop_gpu_prof();

    std::free(xh);
    std::free(yh);
    free_device(x);
    free_device(y);

    return 0;
}

